
#include <hip/hip_runtime.h>
#include <stdio.h>

#define TILE_SIZE 16

__global__ void mysgemm(int m, int n, int k, const float *A, const float *B, float* C) {

    /********************************************************************
     *
     * Compute C = A x B
     *   where A is a (m x k) matrix
     *   where B is a (k x n) matrix
     *   where C is a (m x n) matrix
     *
     * Use shared memory for tiling
     *
     ********************************************************************/

    /*************************************************************************/
    // INSERT KERNEL CODE HERE

    /*************************************************************************/
    __shared__ float ds_A[TILE_SIZE][TILE_SIZE];
    __shared__ float ds_B[TILE_SIZE][TILE_SIZE];

    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    int Row = by * TILE_SIZE + ty;
    int Col = bx * TILE_SIZE + tx;
    float Pvalue = 0;

    for(int i = 0; i < ((k-1) / TILE_SIZE) + 1; i++) {
        if(Row < m && i * TILE_SIZE + tx < k)
                ds_A[ty][tx] = A[Row * k + i * TILE_SIZE + tx];
        else
                ds_A[ty][tx] = 0.0;
        if((i * TILE_SIZE + ty) < k && Col < n)
                ds_B[ty][tx] = B[Col + (i * TILE_SIZE + ty) * n];
        else
                ds_B[ty][tx] = 0.0;
        __syncthreads();
        if(Row < m && Col < n) {
                for(int k = 0; k < TILE_SIZE; ++k)
                        Pvalue += ds_A[ty][k] * ds_B[k][tx];
        }
        __syncthreads();

        }
        if(Row < m && Col < n)
                C[Row * n  + Col] = Pvalue;
}
void basicSgemm(int m, int n, int k, const float *A, const float *B, float *C)
{
    // Initialize thread block and kernel grid dimensions ---------------------

    const unsigned int BLOCK_SIZE = TILE_SIZE;
    const int Width = 1024;

    /*************************************************************************/
    //INSERT CODE HERE

    /*************************************************************************/

    // Invoke CUDA kernel -----------------------------------------------------

    /*************************************************************************/
    //INSERT CODE HERE

    /*************************************************************************/
    dim3 dim_blk(BLOCK_SIZE, BLOCK_SIZE, 1);
    dim3 dim_grid((Width / BLOCK_SIZE), (Width / BLOCK_SIZE), 1);

    mysgemm<<<dim_grid, dim_blk>>>(m, n, k, A, B, C);


}
